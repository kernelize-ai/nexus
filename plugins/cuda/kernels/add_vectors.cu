
#include <hip/hip_runtime.h>

extern "C" __global__ void add_vectors(float* a, float* b, float* c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = a[idx] + b[idx];
}